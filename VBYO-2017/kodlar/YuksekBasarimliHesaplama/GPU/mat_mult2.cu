#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include <stdio.h>
#include <math.h>

#define THREADS_PER_BLOCK 1024
#define TILE_WIDTH 32

void matrixMultiplyCPU(float *a, float *b, float *c, int width) {
  float result;

  for (int row = 0; row < width; row++) {
    for (int col = 0; col < width; col++) {
      result = 0;
      for (int k = 0; k < width; k++) {
        result += a[row * width + k] * b[k * width + col];
      }
      c[row * width + col] = result;
    }
  }
}

void matrixMultiplyCPU_Opt(const float * __restrict a, const float * __restrict__ b, float * __restrict__ c, float * __restrict__ tb, int width) {
  float result;
  
  //transpose b
  for (int row = 0; row < width; row++) {
    for (int col = 0; col < width; col++) {
      tb[col * width + row] = b[row * width + col];
    }
  }

  //can you parallelize this?
  for (int row = 0; row < width; row++) {
    const float* arow = a + row * width;
    for (int col = 0; col < width; col++) {
      const float* bcol = tb + col * width; 

      result = 0;
      for (int k = 0; k < width; k+=8) {
        result += arow[k] * bcol[k];
        result += arow[k+1] * bcol[k+1];
        result += arow[k+2] * bcol[k+2];
        result += arow[k+3] * bcol[k+3];
	result += arow[k+4] * bcol[k+4];
        result += arow[k+5] * bcol[k+5];
        result += arow[k+6] * bcol[k+6];
        result += arow[k+7] * bcol[k+7];
      }
      c[row * width + col] = result;
    }
  }
}

__global__ void matrixMultiplySimple(float *a, float *b, float *c, int width) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  float result = 0;

  if (col < width && row < width) {
    for (int k = 0; k < width; k++) {
      result += a[row * width + k] * b[k * width + col];
    }
    c[row * width + col] = result;
  }
}



__global__ void matrixMultiplyOptimised(float *a, float *b, float *c, int width) {
  // Allocate 2D tiles in shared memory
  __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
  __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

  // Calculate row and column index of element
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float result = 0;
  
  // Loop over tiles of input in phases
  for (int p = 0; p < width / TILE_WIDTH; p++) {
    // Collaboratively load tiles into shared memory
    s_a[threadIdx.y][threadIdx.x] = a[row * width + (p * TILE_WIDTH + threadIdx.x)];
    s_b[threadIdx.y][threadIdx.x] = b[col + width * (p * TILE_WIDTH + threadIdx.y)];

    __syncthreads();     // Wait until all data is loaded before allowing any threads in the block to continue

    // Dot product between row of s_a and column of s_b
    for (int i = 0; i < TILE_WIDTH; i++) {
      result += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
    }
    __syncthreads();    // Wait until all calculations are finished before allowing any threads in the block to continue

  }

  // Write result
  c[row * width + col] = result;
}

int main() {
  int width = 2048; // Define width of square matrix
  // Initialise grid and block variables
  int sqrtThreads = sqrt(THREADS_PER_BLOCK);
  int nBlocks = width/sqrtThreads;
  if (width % sqrtThreads != 0) { // Add an extra block if necessary
    nBlocks++;
  }
  dim3 grid(nBlocks, nBlocks, 1); // this is output data decomposition!!! each thread is responsible from an output entry
  dim3 block(sqrtThreads, sqrtThreads, 1); // Max number of threads per block

  hipSetDevice(1);

  // Initialise host pointers (dynamically allocated memory) and device pointers
  float *a_h;
  float *b_h;
  float *c_h; // GPU results
  float *d_h; // CPU results
  float *a_d;
  float *b_d;
  float *c_d;

  int size; // Number of bytes required by arrays

  // Create timer
  hipEvent_t start;
  hipEvent_t stop;
  float elapsed1, elapsed2, elapsed3;

  // Print out information about blocks and threads
  printf("Number of threads: %i (%ix%i)\n", block.x*block.y, block.x, block.y);
  printf("Number of blocks: %i (%ix%i)\n\n", grid.x*grid.y, grid.x, grid.y);

  // Dynamically allocate host memory
  size = width * width * sizeof(float);
  
  a_h = (float*) malloc(size);
  b_h = (float*) malloc(size);
  float * tb = (float*) malloc(size);
  c_h = (float*) malloc(size);
  d_h = (float*) malloc(size);

  // Load host arrays with data
  for (int i = 0; i < width; i++) {
    for (int j = 0; j < width; j++) {
      a_h[i * width + j] = i + 1.0;
      b_h[i * width + j] = j + 2.0;
      c_h[i * width + j] = 0;
    }
  }

  // Allocate device memory
  cudaCheck(hipMalloc((void**)&a_d, size));
  cudaCheck(hipMalloc((void**)&b_d, size));
  cudaCheck(hipMalloc((void**)&c_d, size));

  // Copy host memory to device memory
  cudaCheck(hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice));

  // Start timer for GPU
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Launch kernel
  matrixMultiplySimple<<<grid, block>>>(a_d, b_d, c_d, width);

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed1, start, stop);

  // Print execution time
  printf("Time to calculate results on GPU: %f ms\n", elapsed1);
  printf("Effective performance: %.3f GFlop\n", ((width/1024.0f) * (width/1024.0f) * (width/1024.0f) * 2.0) / (elapsed1 / 1000));
  printf("Effective bandwith: %.3f GB\n\n", ((3.0 * (width * width) * sizeof(float)) / 1000000000) / (elapsed1 / 1000));
  // Copy results to host
  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  // Start timer for CPU
  hipEventRecord(start, 0);
  matrixMultiplyCPU(a_h, b_h, d_h, width);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed2, start, stop);
  printf("Time to calculate results on CPU: %f ms\n", elapsed2);
  // Compare results
  for (int i = 0; i < width*width; i++) {
    if (c_h[i] != d_h[i]) {
      printf("Error: CPU and GPU results do not match %f %f\n", c_h[i], d_h[i]);
      break;
    }
  }
    
  // Start timer for CPU Opt
  memset(d_h, 0, size);
  hipEventRecord(start, 0);
  matrixMultiplyCPU_Opt(a_h, b_h, d_h, tb, width);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed2, start, stop);
  printf("Time to calculate results on CPU_Opt: %f ms\n\n", elapsed2);
  // Compare results
  for (int i = 0; i < width*width; i++) {
    if (c_h[i] != d_h[i]) {
      printf("Error: CPU and GPU results do not match %f %f\n", c_h[i], d_h[i]);
      break;
    }
  }

  // Start timer for GPU (optimised)
  memset(c_h, 0, size);
  hipMemset(c_d, 0, size);
  hipEventRecord(start, 0);
  matrixMultiplyOptimised<<<grid, block>>>(a_d, b_d, c_d, width);
  cudaCheck(hipPeekAtLastError());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed3, start, stop);
  printf("Time to calculate results on GPU (optimised): %f ms\n", elapsed3);
  printf("Effective performance: %.3f GFlop\n", ((width/1024.0f) * (width/1024.0f) * (width/1024.0f) * 2.0) / (elapsed3 / 1000));
  printf("Effective bandwith: %.3f GB\n\n", ((3.0 * (width/1024.0f * width/1024.0f) * sizeof(float)) / 1024) / (elapsed3 / 1000));

  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  // Compare results
  for (int i = 0; i < width*width; i++) {
    if (c_h[i] != d_h[i]) {
      printf("Error: CPU and GPU (optimised) results do not match %d- %f %f\n", i,c_h[i], d_h[i]);
      break;
    }
  }

  // Free memory
  free(a_h);
  free(b_h);
  free(c_h);
  free(d_h);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
