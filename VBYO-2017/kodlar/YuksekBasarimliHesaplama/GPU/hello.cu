
#include <hip/hip_runtime.h>
#include <stdio.h>

//__device__ const char *STR = "HELLO WORLD! ";
__device__ const char *STR = "HELLO WORLD! HELLO WORLD! HELLO WORLD! HELLO WORLD! ";

//const char STR_LENGTH = 13;
const char STR_LENGTH = 52;

__global__ void hello() {
  printf("%c", STR[threadIdx.x % STR_LENGTH]);
}

int main(void) {
  int num_threads = STR_LENGTH;
  int num_blocks = 1;
  
  hello<<<num_blocks,num_threads>>>();
  hipDeviceSynchronize();		
  
  return 0;
}
