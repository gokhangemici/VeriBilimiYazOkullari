#include "hip/hip_runtime.h"
//compile with nvcc -O3 -arch=sm_35

#include <stdio.h>
#include "common.h"

#define TIMES 1024
#define N (1024 * 1024)

__global__ void dummy(int *a, int *b) {
  int index = blockDim.x * threadIdx.x + blockIdx.x;
  
  int sum = 0;
  for(int i = 0; i < TIMES; i++) {
    sum += a[(index + i) % N];
  }

  b[index] = sum;
}

//dummy kernel; each thread adds some portion of a and store it in b
__global__ void dummy2(int *a, int *b) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  int sum = 0;
  for(int i = 0; i < TIMES; i++) {
    sum += a[(index + i) % N];
  }

  b[index] = sum;
}

int main() {
  /******************************************************************************/
  //Preparing the memory
  int *a, *b;
  int *d_a, *d_b;
  size_t size = N * sizeof( int );
  
  cudaCheck(hipMalloc( (void **) &d_a, size ));
  cudaCheck(hipMalloc( (void **) &d_b, size ));
  
  /* allocate space for host copies of a, b, c and setup input values */
  a = (int *)malloc( size );
  b = (int *)malloc( size );
  
  for(int i = 0; i < N; i++ ) {
    a[i] = 1;
    b[i] = 0;
  }
  /******************************************************************************/
  
  /******************************************************************************/
  //Timing
  hipEvent_t start,stop;
  float elapsedTime;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start,0);
  /* copy inputs to device */
  /* fix the parameters needed to copy data to the device */
  cudaCheck(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
  cudaCheck(hipMemset(d_b, 0, size));
  /* launch the kernel on the GPU */
  /* insert the launch parameters to launch the kernel properly using blocks and threads */
  dummy<<<1024, 1024>>>(d_a, d_b);
  
  /* copy result back to host */
  /* fix the parameters needed to copy data back to the host */
  hipDeviceSynchronize();
  cudaCheck(hipPeekAtLastError());
  cudaCheck(hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost ));
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  /******************************************************************************/
  for( int i = 0; i < N; i++) {
    if(b[i] != TIMES) {
      printf("GPU Error: value b[%d] = %d\n", i, b[i]);
      break;
    }
  }
  printf("GPU time is: %lf seconds\n", elapsedTime / 1000);
  
  hipEventRecord(start,0);
  /* copy inputs to device */
  /* fix the parameters needed to copy data to the device */
  cudaCheck(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
  cudaCheck(hipMemset(d_b, 0, size));
  /* launch the kernel on the GPU */
  /* insert the launch parameters to launch the kernel properly using blocks and threads */
  dummy2<<<1024, 1024>>>(d_a, d_b);

  /* copy result back to host */
  /* fix the parameters needed to copy data back to the host */
  hipDeviceSynchronize();
  cudaCheck(hipPeekAtLastError());
  cudaCheck(hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost ));
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  /******************************************************************************/
  for( int i = 0; i < N; i++) {
    if(b[i] != TIMES) {
      printf("GPU Error: value b[%d] = %d\n", i, b[i]);
      break;
    }
  }
  printf("GPU time is: %lf seconds\n", elapsedTime / 1000);


  /* clean up */
  free(a);
  free(b);
  hipFree( d_a );
  hipFree( d_b );
  
  return 0;
} /* end main */
